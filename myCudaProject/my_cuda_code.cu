
#include <hip/hip_runtime.h>
__global__ void addArrays(float* a, float* b, float* result, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        result[tid] = a[tid] + b[tid];
    }
}
